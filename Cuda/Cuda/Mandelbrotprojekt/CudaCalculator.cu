
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <sstream>
#include <stdio.h>
#include <iostream>
#include "Client.h"

using namespace std;

class CudaCalculator {

public:
	CudaCalculator(Client &client, int imageWidth, int imageHeight);
	void calculate(int y, double xMove, double yMove, double zoom, int itr);
	void getTask();
	void formatResult(int x, int y, int itr);

	int getIntOfCharArray(char arr[]);
	double getDoubleOfCharArray(char arr[]);

private:
	Client *client;
	string package;
	int imageWidth;
	int imageHeight;
	int len = 0;
	double zy = 0;
	double zx = 0;
	double cx = 0;
	double cy = 0;
	double temp = 0;
	char check[128];
	char y_bytes[4];
	char xMove_bytes[8];
	char yMove_bytes[8];
	char zoom_bytes[8];
	char itr_bytes[4];
};


CudaCalculator::CudaCalculator(Client &client, int imageWidth, int imageHeight) {
	this->client = &client;
	this->imageWidth = imageWidth;
	this->imageHeight = imageHeight;
}

void CudaCalculator::calculate(int y, double xMove, double yMove, double zoom, int itr) {

	double tmp;

		for (int x = 0; x < imageWidth; x++) {
			zx = 0;
			zy = 0;
			cx = ((double) x - (imageWidth / 2.0) + xMove) / zoom;
			cy = ((double) y - (imageHeight / 2.0) + yMove) / zoom;
			tmp = itr;

			while ((zx * zx + zy * zy) < 4.0 && tmp > 0) {
				temp = zx * zx - zy * zy + cx;
				zy = 2 * zx * zy + cy;
				zx = temp;
				tmp -= 1;
			}
			formatResult(x, y, tmp);
		}
			client->sendMessage(package.c_str());
			package.clear();

	client->sendMessage("tick\n");
}

void CudaCalculator::formatResult(int x, int y, int itr) {

//	package.append("\n").append(to_string(y)).append("\n").append(to_string(itr)).append("\n");
	package += to_string(x);
	package	+= "\n";
	package	+= to_string(y);
	package	+= "\n";
	package	+= to_string(itr);
	package	+= "\n";

}


void CudaCalculator::getTask() {

	do {
		client->sendMessage("task\n");
		client->receiveMessage(check, 128);
	} while (strcmp(check, "noTask") == 0);

	if(strcmp(check, "task") == 0) {

		client->receiveMessage(y_bytes, 4);
		client->receiveMessage(xMove_bytes, 8);
		client->receiveMessage(yMove_bytes, 8);
		client->receiveMessage(zoom_bytes, 8);
		client->receiveMessage(itr_bytes, 8);


		int y = getIntOfCharArray(y_bytes);
		double xMove = getDoubleOfCharArray(xMove_bytes);
		double yMove = getDoubleOfCharArray(yMove_bytes);
		double zoom = getDoubleOfCharArray(zoom_bytes);
		int itr = getIntOfCharArray(itr_bytes);

		calculate(y, xMove, yMove, zoom, itr);
	}
}

int CudaCalculator:: getIntOfCharArray(char arr[]){

	unsigned char* res = (unsigned char*)arr;
	int result = (int)(res[0] | res[1] << 8 | res[2] << 16 | res[3] << 24);
	return result;
}

double CudaCalculator::getDoubleOfCharArray(char arr[]) {

	unsigned char* res = (unsigned char*)(arr);
	double result = *reinterpret_cast<double*>(res);
	return result;
}
