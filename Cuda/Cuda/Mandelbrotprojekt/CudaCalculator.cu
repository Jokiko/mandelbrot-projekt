
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Client.h"

class CudaCalculator {

public:
	CudaCalculator(Client client, int imageWidth, int imageHeight);
	void calculate(int iteration);
	void cudaCalculate(int iteration);

private:
	Client client;
	int imageWidth;
	int imageHeight;
	double zy = 0;
	double zx = 0;
	double cx = 0;
	double cy = 0;
	double temp = 0;
	int itr = 0;


};


CudaCalculator::CudaCalculator(Client client, int imageWidth, int imageHeight) {
	this->client = client;
	this->imageWidth = imageWidth;
	this->imageHeight = imageHeight;
}

void CudaCalculator::calculate(int iteration) {

	for (int y = 0; y < imageHeight; y++) {
		for (int x = 0; x < imageWidth; x++) {
			zx = zy = 0;
			cx = x - (imageWidth / 2.0);
			cy = y - (imageHeight / 2.0);
			itr = iteration;

			while (zx * zx + zy * zy < 4 && itr > 0) {
				temp = zx * zx - zy * zy + cx;
				zy = 2 * zx * zy + cy;
				zx = temp;
				itr--;
			}
		}
	}
}

