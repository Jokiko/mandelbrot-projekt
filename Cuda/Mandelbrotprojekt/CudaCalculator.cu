﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <time.h>

__device__ double zx;
__device__ double zy;
__device__ double cx;
__device__ double cy;
__device__ double temp;
__device__ int offset;

/*
Naive Bestimmung der Anzahl an Ziffern einer Integer. ;)
*/
__device__ int getNumberOfDigits(int n) {

	if (n >= 1000000000)
		return 10;

	if (n >= 100000000)
		return 9;

	if (n >= 10000000)
		return 8;

	if (n >= 1000000)
		return 7;

	if (n >= 100000)
		return 6;

	if (n >= 10000)
		return 5;

	if (n >= 1000)
		return 4;

	if (n >= 100)
		return 3;

	if (n >= 10)
		return 2;

	return 1;
}

/*
Da der Server Strings empfängt, muss ein ordentlich formatierter ASCII String mühselig 
selbst erzeug werden. CUDA __device__ code kann keine std::string Methoden oder
ähnliches verwenden. :( 
*/

__device__ void formatResult(int x, int y, int itr, char* package) {

	int tmp_x = x;
	int tmp_y = y;
	int tmp_itr = itr;

	int x_digits = 0;
	int y_digits = 0;
	int itr_digits = 0;

	int x_digits_tmp = 0;
	int y_digits_tmp = 0;
	int itr_digits_tmp = 0;

	if (tmp_x == 0) {
		*(package + offset++) = '0';
		*(package + offset++) = '\n';
	}
	else {

		x_digits = getNumberOfDigits(tmp_x);
		x_digits_tmp = x_digits - 1;

		while (x_digits_tmp >= 0) {

			/*
			* Die erste Ziffer einer Zahl wird als letztes berechnet, und umgekehrt.
			* Die zuletzt berechnete Zahl muss also als erstes geschrieben werden,
			* daher der zusätzliche offset "x_digits_tmp".
			*/
			*(package + x_digits_tmp-- + offset) = (tmp_x % 10) + '0';
			tmp_x /= 10;
		}

		offset += x_digits;

		*(package + offset++) = '\n';
	}

	if (tmp_y == 0) {
		*(package + offset++) = '0';
		*(package + offset++) = '\n';
	}
	else {

		y_digits = getNumberOfDigits(tmp_y);
		y_digits_tmp = y_digits - 1;

		while (y_digits_tmp >= 0) {
			*(package + y_digits_tmp-- + offset) = tmp_y % 10 + '0';
			tmp_y /= 10;
		}

		offset += y_digits;
		*(package + offset++) = '\n';
	}

	if (tmp_itr == 0) {
		*(package + offset++) = '0';
		*(package + offset++) = '\n';
	}
	else {

		itr_digits = getNumberOfDigits(tmp_itr);
		itr_digits_tmp = itr_digits - 1;
		while (itr_digits_tmp >= 0) {
			*(package + itr_digits_tmp-- + offset) = tmp_itr % 10 + '0';
			tmp_itr /= 10;
		}
		offset += itr_digits;
		*(package + offset++) = '\n';
	}

}

__global__ void calculate(int imageWidth, int imageHeight, int y, double xMove, double yMove, double zoom, int itr, char* package) {

	int tmp_itr;

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	offset = 0;

	for (int x = index; x < imageWidth; x += stride) {

		zx = 0;
		zy = 0;
		cx = ((double)x - (imageWidth / 2.0) + xMove) / zoom;
		cy = ((double)y - (imageHeight / 2.0) + yMove) / zoom;

		tmp_itr = itr;

		while ((zx * zx + zy * zy) < 4.0 && tmp_itr > 0) {
			temp = zx * zx - zy * zy + cx;
			zy = 2 * zx * zy + cy;
			zx = temp;
			tmp_itr -= 1;
		}
		formatResult(x, y, tmp_itr, package);
	}

	package[offset] = '\0';

}

