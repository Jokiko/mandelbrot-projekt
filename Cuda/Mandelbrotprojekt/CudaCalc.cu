﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__device__ double zx;
__device__ double zy;
__device__ double cx;
__device__ double cy;
__device__ double temp;
__device__ int offset;

__device__ void formatResult(int x, int y, int itr, char* package) {

	//printf("x: %d ; y: %d; itr: %d\n", x, y, itr);

	char* tmp_x = (char*)&x;
	char* tmp_y = (char*)&y;
	char* tmp_itr = (char*)&itr;

	for (int i = 3; i >= 0; i--) {
		*(package + offset++) = tmp_x[i];
	}

	package[offset++] = '\n';

	for (int i = 3; i >= 0; i--) {
		package[offset++] = tmp_y[i];
	}

	package[offset++] = '\n';

	for (int i = 0; i < 4; i++) {

		package[offset++] = tmp_itr[i];
	}

	package[offset++] = '\n';
}

__global__ void calculate(int imageWidth, int imageHeight, int y, double xMove, double yMove, double zoom, int itr, char* package) {

	int tmp_itr;

	offset = 0;

	for (int x = 0; x < imageWidth; x++) {
		zx = 0;
		zy = 0;
		cx = ((double)x - (imageWidth / 2.0) + xMove) / zoom;
		cy = ((double)y - (imageHeight / 2.0) + yMove) / zoom;
		tmp_itr = itr;

		while ((zx * zx + zy * zy) < 4.0 && tmp_itr > 0) {
			temp = zx * zx - zy * zy + cx;
			zy = 2 * zx * zy + cy;
			zx = temp;
			tmp_itr -= 1;
		}
		formatResult(x, y, tmp_itr, package);
	}

	package[offset] = '\0';
}

