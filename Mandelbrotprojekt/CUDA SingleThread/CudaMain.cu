﻿#include "CudaCalculator.cuh"
#include "Client.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <windows.h>
#include <time.h>

#pragma comment( lib, "ws2_32.lib" )

using namespace std;

Client client;
char input_buffer[256];
int image_width, image_height;
int result_package_size;

char* host_result_package;
char* device_result_package;

int task_y;
double task_xMove;
double task_yMove;
double task_zoom;
int task_itr;



//Connect to Server
void connect() {
	client.sendMessage("connect/.../\n");
	client.receiveMessage(input_buffer, 256);
	printf(input_buffer);
}

//Request image resolution
void receiveSize() {

	client.sendMessage("width\n");
	client.receiveMessage(input_buffer, 256);
	image_width = *(int*)input_buffer;

	client.sendMessage("height\n");
	client.receiveMessage(input_buffer, 256);
	image_height = *(int*)input_buffer;

	result_package_size = (3 * (sizeof(int) + 1) * image_width) + 1;

}

void setupHostVariables() {
	host_result_package = (char*)malloc(result_package_size);
}

void setupDeviceVariables() {
	hipMalloc(&device_result_package, result_package_size);
}

void copyResults() {
	hipMemcpy(host_result_package, device_result_package, result_package_size, hipMemcpyDeviceToHost);
}

void getTask() {

	char check[128];
	char y_bytes[4];
	char xMove_bytes[8];
	char yMove_bytes[8];
	char zoom_bytes[8];
	char itr_bytes[4];

	client.sendMessage("task\n");
	client.receiveMessage(check, 128);

	if (strcmp(check, "task") == 0) {

		client.sendMessage("s\n");
		client.receiveMessage(y_bytes, 4);
		client.sendMessage("s\n");
		client.receiveMessage(xMove_bytes, 8);
		client.sendMessage("s\n");
		client.receiveMessage(yMove_bytes, 8);
		client.sendMessage("s\n");
		client.receiveMessage(zoom_bytes, 8);
		client.sendMessage("s\n");
		client.receiveMessage(itr_bytes, 4);

		task_y = *(int*)y_bytes;
		task_xMove = *(double*)xMove_bytes;
		task_yMove = *(double*)yMove_bytes;
		task_zoom = *(double*)zoom_bytes;
		task_itr = *(int*)itr_bytes;



		calculate << <1, 1 >> > (image_width, image_height, task_y, task_xMove, task_yMove, task_zoom, task_itr, device_result_package);
		hipDeviceSynchronize();
		copyResults();

		client.sendMessage(host_result_package);
		client.sendMessage("tick\n");

		ZeroMemory(check, 128);
		ZeroMemory(y_bytes, 4);
		ZeroMemory(xMove_bytes, 8);
		ZeroMemory(yMove_bytes, 8);
		ZeroMemory(zoom_bytes, 8);
		ZeroMemory(itr_bytes, 4);

	}
	else {
		client.sendMessage("tick\n");
	}
}

int main(int argc, char const* argv[])
{


	connect();
	receiveSize();
	setupHostVariables();
	setupDeviceVariables();

	while (true) {
		getTask();
	};
}



