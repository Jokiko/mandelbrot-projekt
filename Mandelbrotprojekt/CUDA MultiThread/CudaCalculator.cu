﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <time.h>

/*
Naive Bestimmung der Anzahl an Ziffern einer Integer. ;)
*/
__device__ int getNumberOfDigits(int n) {

	if (n >= 1000000000)
		return 10;

	if (n >= 100000000)
		return 9;

	if (n >= 10000000)
		return 8;

	if (n >= 1000000)
		return 7;

	if (n >= 100000)
		return 6;

	if (n >= 10000)
		return 5;

	if (n >= 1000)
		return 4;

	if (n >= 100)
		return 3;

	if (n >= 10)
		return 2;

	return 1;
}

__global__ void calculate(int imageWidth, int imageHeight, int y, double xMove, double yMove, double zoom, int itr, char* package) {

	int tmp_x = threadIdx.x;
	int tmp_y = y;
	int tmp_itr = itr;

	int x_digits = 0;
	int y_digits = 0;
	int itr_digits = 0;

	int x_digits_tmp = 0;
	int y_digits_tmp = 0;
	int itr_digits_tmp = 0;

	int offset = threadIdx.x * 15;

	double zx = 0;
	double zy = 0;
	double cx = ((double)tmp_x - (imageWidth / 2.0) + xMove) / zoom;
	double cy = ((double)y - (imageHeight / 2.0) + yMove) / zoom;
	double temp = 0;

	while ((zx * zx + zy * zy) < 4.0 && tmp_itr > 0) {
		temp = zx * zx - zy * zy + cx;
		zy = 2 * zx * zy + cy;
		zx = temp;
		tmp_itr -= 1;
	}

	/*
	Da der Server Strings empfängt, muss ein ordentlich formatierter ASCII String mühselig
	selbst erzeug werden. CUDA __device__ code kann keine std::string Methoden oder
	ähnliches verwenden. :(
	*/

	if (tmp_x == 0) {
		*(package + offset++) = '0';
		*(package + offset++) = '\n';
	}
	else {
		x_digits = getNumberOfDigits(tmp_x);
		x_digits_tmp = x_digits - 1;

		while (x_digits_tmp >= 0) {

			/*
			* Die erste Ziffer einer Zahl wird als letztes berechnet, und umgekehrt.
			* Die zuletzt berechnete Zahl muss also als erstes geschrieben werden,
			* daher der zusätzliche offset "x_digits_tmp".
			*/
			*(package + x_digits_tmp-- + offset) = (tmp_x % 10) + '0';
			tmp_x /= 10;
		}

		offset += x_digits;

		*(package + offset++) = '\n';
	}

	if (tmp_y == 0) {
		*(package + offset++) = '0';
		*(package + offset++) = '\n';
	}
	else {

		y_digits = getNumberOfDigits(tmp_y);
		y_digits_tmp = y_digits - 1;

		while (y_digits_tmp >= 0) {
			*(package + y_digits_tmp-- + offset) = tmp_y % 10 + '0';
			tmp_y /= 10;
		}

		offset += y_digits;
		*(package + offset++) = '\n';
	}

	if (tmp_itr == 0) {
		*(package + offset++) = '0';
		*(package + offset++) = '\n';
	}
	else {

		itr_digits = getNumberOfDigits(tmp_itr);
		itr_digits_tmp = itr_digits - 1;
		while (itr_digits_tmp >= 0) {
			*(package + itr_digits_tmp-- + offset) = tmp_itr % 10 + '0';
			tmp_itr /= 10;
		}
		offset += itr_digits;
		*(package + offset++) = '\n';
	}

	if (threadIdx.x == imageWidth - 1) {
		package[offset] = '\0';
	}
	else {
		while (offset < (threadIdx.x + 1) * 15)
			package[offset++] = 0x20;
	}
}

